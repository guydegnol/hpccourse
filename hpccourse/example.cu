#include "hip/hip_runtime.h"
%%ipsa_nvcudac_and_exec
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <iostream>

using namespace std;


__global__ void kernel_func(int *a) {
    unsigned int thread_idx = threadIdx.x + blockDim.x * blockIdx.x;
    ...
}

int main (int argc, char *argv[]) {

  // Do sequential stuff
  malloc(...);
  ...  

  // Allocate object on the Device
  hipMalloc(...);

  // Copy memory from Host to Device
  hipMemcpy(..., hipMemcpyHostToDevice);

  // Launch Kernel function
  kernel_func<<<grid_size, block_size>>>(...);

  // Copy memory from Device to Host
  hipMemcpy(..., hipMemcpyDeviceToHost);

  // Deallocate object on the Device
  hipFree(...);

  // Do other sequential stuff
  ...  
  free(...);
}